#include <thrust/device_vector.h>
#include <iostream>
#include "../COMMON/commons.cuh"
using namespace std;
int main(int argc, char **argv){

    thrust::device_vector<float> v(10, 2);
    ofstream outFile;
    outFile.open("./output", ios::out);
    thrust::copy(v.begin(), v.end(), std::ostream_iterator<float>(outFile, " "));
    CUDA_CHECK(hipDeviceSynchronize());
    outFile.close();
return 0;
}
