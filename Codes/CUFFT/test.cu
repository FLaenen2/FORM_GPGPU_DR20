#include <thrust/device_vector.h>
#include <iostream>
#include "../COMMON/commons.cuh"
using namespace std;
int main(int argc, char **argv){

    thrust::device_vector<float> v(10, 2);
    ifstream inFile;
    inFile.open("./outputHeat", ios::in);
    istream_iterator<float> intvecRead ( inFile );
    thrust::copy(intvecRead, std::istream_iterator<float>(), std::ostream_iterator<float>(std::cout, " "));
    CUDA_CHECK(hipDeviceSynchronize());
    inFile.close();
return 0;
}
