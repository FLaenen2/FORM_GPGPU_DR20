#include "hip/hip_runtime.h"
#include <iostream> 
#include <stdio.h>
#include <hipfft/hipfft.h>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <iostream>
#include "../COMMON/commons.cuh"


template<class C>
__global__ void scale_cmp(C *arr, const float scale, const int size){

    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < size){
        arr[i].x *= scale;
        arr[i].y *= scale;
    }
}


template<typename T>
    __global__ void printme(T *val, int size = 1){

        for (int i = 0; i < size; i++){
            printf("%g\t", val[i]);
        }
        printf("\n");
    }


template<typename C>
    __global__ void print_cmp(C *val, int size = 1){

        for (int i = 0; i < size; i++){
            printf("[%d] %g\t", i, val[i].x * val[i].x + val[i].y * val[i].y);
        }
        printf("\n");
    }



#define TPB 512

__global__ void update(hipfftComplex *vec, const float nu, const float dt, const int nk){
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < nk){
	int kx = i;
	float kk = kx * kx;	
	vec[i].x -= dt * nu * kk * vec[i].x;
	vec[i].y -= dt * nu * kk * vec[i].y;
	//vec[i].x *= (1 - nu * dt * kk ) * vec[i].x;
	//vec[i].y *= (1 - nu * dt * kk ) * vec[i].y;
    }

}

using namespace std;
                        ///////////
                        /// MAIN //

int main(int argc, char **argv){
	
    int nx = 1024;
    int nk = nx/2 + 1;
    float L = 2. * M_PI;
    float *h_v = new float[nx]();
    hipfftComplex *d_v;

    ofstream outFile;
    outFile.open("./outputHeat", ios::out);
    outFile.close();
    size_t spatSize = nx * sizeof(float);     // Shortcut for memory copies and allocations
    size_t specSize = nk * sizeof(hipfftComplex);
    
    // Parameters
    float nu = .005;
    float T  = 1;
    float dt = 0.001;
    int NT = round(T / dt);
    float dx = L / nx;
    float mean = 0;
    float sigma = 0.5;
    // Generate a signal on the host
    for (int i = 0; i < nx; i++){
	    float xi = -M_PI + i * dx;
	    h_v[i] = exp(-(xi-mean)*(xi-mean)/(2.*sigma*sigma)); 
     }
    

    CUDA_CHECK(hipMalloc(&d_v, specSize));
    // Pointer needs to be converted to hipfftReal when necessary
    CUDA_CHECK(hipMemset((hipfftReal *)d_v, 0, specSize));
    CUDA_CHECK(hipMemcpy((hipfftReal *)d_v, h_v, spatSize, H2D));
    
    // Create plans
    hipfftHandle planr2c;    // the names r2c and c2r will be used regardless of the precison
    hipfftHandle planc2r;
    CUFFT_CHECK(hipfftPlan1d(&planr2c, nx, HIPFFT_R2C, 1));
    CUFFT_CHECK(hipfftPlan1d(&planc2r, nx, HIPFFT_C2R, 1));

		// for no padding : useless in 1D
    CUFFT_CHECK(cufftSetCompatibilityMode(planr2c, CUFFT_COMPATIBILITY_NATIVE));
    CUFFT_CHECK(cufftSetCompatibilityMode(planc2r, CUFFT_COMPATIBILITY_NATIVE));

    int grid (ceil((float)nk / TPB));
    outFile.open("./outputHeat", ios::app);
    thrust::device_ptr<hipfftReal> dev_ptr((hipfftReal *)d_v);
    thrust::copy(dev_ptr, dev_ptr + nx, ostream_iterator<hipfftReal>(outFile, " "));
    CUDA_CHECK(hipDeviceSynchronize());
    outFile << endl;
    outFile.close();

    // DIRECT TRANSFORM, inplace
    CUFFT_CHECK(hipfftExecR2C(planr2c, (hipfftReal *)d_v,  d_v));
    scale_cmp<<<grid, TPB>>>(d_v, 1./nx, nk);
    CUDA_CHECK_ERROR();
    
    for (int it = 0; it < NT ; it++){
	
	update<<<grid, TPB>>>(d_v, nu, dt, nk);     
	CUDA_CHECK_ERROR();
	//CUDA_CHECK(hipDeviceSynchronize());
	
	// INVERSE TRANSFORM
	    if (!(it % 10)){
		CUFFT_CHECK(hipfftExecC2R(planc2r, d_v, (hipfftReal *)d_v));
		CUDA_CHECK(hipDeviceSynchronize());
		outFile.open("./outputHeat", ios::app);
		thrust::device_ptr<hipfftReal> dev_ptr((hipfftReal *)d_v);
		thrust::copy(dev_ptr, dev_ptr + nx, ostream_iterator<hipfftReal>(outFile, " "));
		CUDA_CHECK(hipDeviceSynchronize());
		outFile << endl;
		outFile.close();
		CUFFT_CHECK(hipfftExecR2C(planr2c, (hipfftReal *)d_v,  d_v));
		scale_cmp<<<grid, TPB>>>(d_v, 1./nx, nk);
		CUDA_CHECK_ERROR();
	    } 
    }

    // Release plans
    CUFFT_CHECK(hipfftDestroy(planr2c));
    CUFFT_CHECK(hipfftDestroy(planc2r));
	
    return 0;
    
}




