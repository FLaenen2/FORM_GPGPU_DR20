#include "hip/hip_runtime.h"
#include <iostream> 
#include <stdio.h>
#include <hipfft/hipfft.h>
#include "../COMMON/commons.cuh"


 template<class C>
    __global__ void compLap(const C *src, C *dst, const int nx, const int ny, const float dkx = 1, const float dky = 1){
    
        int i = blockIdx.x * blockDim.x + threadIdx.x;
        int j = blockIdx.y * blockDim.y + threadIdx.y;
        if (i < (nx/2+1) && j < ny){
            int gidx = j * (nx/2+1) + i;
            j = (j > ny/2) ? (j - ny) : j;
            float kx = i*dkx;
            float ky = j*dky;
            float kk = (float)(kx*kx + ky*ky);
            dst[gidx].x = -kk * src[gidx].x;
            dst[gidx].y = -kk * src[gidx].y;
        }    
    }
    
template<class C>
__global__ void scale_cmp(C *arr, const float scale, const int size){

    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < size){
        arr[i].x *= scale;
        arr[i].y *= scale;
    }

}


template<typename T>
    __global__ void printme(T *val, int size = 1){

        for (int i = 0; i < size; i++){
            printf("%g\t", val[i]);
        }
        printf("\n");

    }


template<typename C>
    __global__ void print_cmp(C *val, int size = 1){

        for (int i = 0; i < size; i++){
            printf("%g\t", val[i].x * val[i].x + val[i].y * val[i].y);
        }
        printf("\n");

    }



#define TPB 512

using namespace std;
                        ///////////
                        /// MAIN //

int main(int argc, char **argv){

	CUDA_CHECK(hipDeviceReset());
	
    int nx = 16;
    int ny = 16;
    int nk = ny * (nx/2 + 1);
    float L = 2. * M_PI;
    float dx = L / nx, dy = L / ny;
    float *h_v = new float[nx * ny]();
    hipfftComplex *d_v;

    size_t spatSize = nx * ny * sizeof(float);     // Shortcut for memory copies and allocations
    size_t specSize = nk * sizeof(hipfftComplex);
    
    // Generate a signal on the host
    for (int i = 0; i < ny; i++){
        for (int j = 0; j < nx; j++){
	    	h_v[i*nx + j] = sin(j * dx) * cos(i * dy) ; 
	    }
     }

    CUDA_CHECK(hipMalloc(&d_v, specSize));
    CUDA_CHECK(hipMemset(d_v, 0, specSize));
    CUDA_CHECK(hipMemcpy((hipfftReal *) d_v, h_v, spatSize, H2D));
    cout << endl << "Before : " << endl;
    printme<<<1,1>>>((hipfftReal *) d_v, nx);
    CUDA_CHECK_ERROR();
    CUDA_CHECK(hipDeviceSynchronize());
    
    // Create plans
    hipfftHandle planr2c;    // the names r2c and c2r will be used regardless of the precison
    hipfftHandle planc2r;
	CUFFT_CHECK(hipfftPlan2d(&planr2c, ny, nx, HIPFFT_R2C));
    CUFFT_CHECK(hipfftPlan2d(&planc2r, ny, nx, HIPFFT_C2R));
	SYNCGPU();
		// for no padding
    CUFFT_CHECK(cufftSetCompatibilityMode(planr2c, CUFFT_COMPATIBILITY_NATIVE));
    CUFFT_CHECK(cufftSetCompatibilityMode(planc2r, CUFFT_COMPATIBILITY_NATIVE));
	SYNCGPU();
    int grid (ceil((float)nk / TPB));

    // DIRECT TRANSFORM
    CUFFT_CHECK(hipfftExecR2C(planr2c, (hipfftReal *) d_v,  d_v));
    scale_cmp<<<grid, TPB>>>(d_v, 1./(nx*ny), nk);
    CUDA_CHECK_ERROR();
    CUDA_CHECK(hipDeviceSynchronize());
    int tpbx = 16;
    int tpby = 16;
    dim3 block2D(tpbx, tpby);
    dim3 grid2D(ceil((float)nx/2+1/tpbx), ceil((float)ny/tpby));
    compLap<<<grid2D, block2D>>>(d_v, d_v, nx, ny);
    CUDA_CHECK_ERROR();
    CUDA_CHECK(hipDeviceSynchronize());
    
    // INVERSE TRANSFORM
    CUFFT_CHECK(hipfftExecC2R(planc2r, d_v, (hipfftReal *) d_v));
        
    // PRINT AFTER
    cout << endl << "After : " << endl;
    printme<<<1,1>>>((hipfftReal *) d_v, nx);
    CUDA_CHECK_ERROR();
    CUDA_CHECK(hipDeviceSynchronize());

	// Release plans
	CUFFT_CHECK(hipfftDestroy(planr2c));
	CUFFT_CHECK(hipfftDestroy(planc2r));
	
    return 0;
    
}




