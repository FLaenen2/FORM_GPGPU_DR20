#include "hip/hip_runtime.h"
#include <iostream> 
#include <stdio.h>
#include <hipfft/hipfft.h>
#include "../COMMON/commons.cuh"
#include <hip/hip_complex.h>

template<class C>
__global__ void scale_cmp(C *arr, const float scale, const int size){

    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < size){
        arr[i].x *= scale;
        arr[i].y *= scale;
    }

}


template<typename T>
    __global__ void printme(T *val, int size = 1){

        for (int i = 0; i < size; i++){
            printf("%g\t", val[i]);
        }
        printf("\n");

    }


template<typename C>
    __global__ void print_cmp(C *val, int size = 1){

        for (int i = 0; i < size; i++){
            //printf("[%d] %g\t", i, val[i].x * val[i].x + val[i].y * val[i].y);
            printf("[%d] %g\t", i, hipCabsf(val[i]) * hipCabsf(val[i]));
        }
        printf("\n");

    }



#define TPB 512

using namespace std;
                        ///////////
                        /// MAIN //

int main(int argc, char **argv){
	
    int nx = 64;
    int f1 = 5;  // Frequencies
    int f2 = 10;
    int nk = nx/2 + 1;
    float L = 2. * M_PI;
    float *h_v = new float[nx]();
    hipfftComplex *d_v;

    size_t spatSize = nx * sizeof(float);     // Shortcut for memory copies and allocations
    size_t specSize = nk * sizeof(hipfftComplex);
    
    // Generate a signal on the host
    for (int i = 0; i < nx; i++){
	    h_v[i] = cos((float) f1 * i / nx * L) + 2 * cos((float) f2 * i / nx * L) ; 
     }
    

    CUDA_CHECK(hipMalloc(&d_v, specSize));
    // Pointer needs to be converted to hipfftReal when necessary
    CUDA_CHECK(hipMemset((hipfftReal *)d_v, 0, specSize));
    CUDA_CHECK(hipMemcpy((hipfftReal *)d_v, h_v, spatSize, H2D));
    cout << endl << "Before : " << endl;
    printme<<<1,1>>>((hipfftReal *) d_v, nx);
    CUDA_CHECK_ERROR();
    CUDA_CHECK(hipDeviceSynchronize());
    
    // Create plans
    hipfftHandle planr2c;    // the names r2c and c2r will be used regardless of the precison
    hipfftHandle planc2r;
    CUFFT_CHECK(hipfftPlan1d(&planr2c, nx, HIPFFT_R2C, 1));
    CUFFT_CHECK(hipfftPlan1d(&planc2r, nx, HIPFFT_C2R, 1));

		// for no padding
    CUFFT_CHECK(cufftSetCompatibilityMode(planr2c, CUFFT_COMPATIBILITY_NATIVE));
    CUFFT_CHECK(cufftSetCompatibilityMode(planc2r, CUFFT_COMPATIBILITY_NATIVE));


	// Illustrates the use of timer
    GpuTimer myT(1);

    myT.Start();
    // DIRECT TRANSFORM, inplace
    CUFFT_CHECK(hipfftExecR2C(planr2c, (hipfftReal *) d_v,  d_v));
    int grid (ceil((float)nk / TPB));
    scale_cmp<<<grid, TPB>>>(d_v, 1./nx, nk);
    CUDA_CHECK_ERROR();
    cout << endl << "Power spectrum : " << endl;
    print_cmp<<<1, 1>>>(d_v, nk);
    CUDA_CHECK_ERROR();
    CUDA_CHECK(hipDeviceSynchronize());
    myT.Stop();
    cout << "Time elapsed " << myT.Elapsed() << endl;

    // INVERSE TRANSFORM
    CUFFT_CHECK(hipfftExecC2R(planc2r, d_v, (hipfftReal *)d_v));
    
    // PRINT AFTER
    cout << endl << "After : " << endl;
    printme<<<1,1>>>((hipfftReal *) d_v, nx);
    CUDA_CHECK_ERROR();
    CUDA_CHECK(hipDeviceSynchronize());

    // Release plans
    CUFFT_CHECK(hipfftDestroy(planr2c));
    CUFFT_CHECK(hipfftDestroy(planc2r));
	
    return 0;
    
}




