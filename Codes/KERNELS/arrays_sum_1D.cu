#include "hip/hip_runtime.h"
#include "../COMMON/commons.cuh"

#define H2D hipMemcpyHostToDevice
#define D2H hipMemcpyDeviceToHost

template<typename T>
__global__ void sum(const T *S1, const T *S2, T *S3, const int n){

	int i = blockIdx.x * blockDim.x + threadIdx.x; 
	if (i < n) { // or if (i >= n){ return ;}
		S3[i] = S1[i] + S2[i];
	}

}


int main (int argc, char **argv){

	int n = 100;
	
	// Make memory space for host arrays, optionally instanciated to zero
	int *h_S1 = new int[n]();
	int *h_S2 = new int[n]();
	
	// Fill CPU arrays;
	for (int i = 0; i < n; i++){
		h_S1[i] = i;
		h_S2[i] = 2 * i;
	}
	
	// Declare pointers for memory arrays
	int *d_S1, *d_S2, *d_S3;
	
	// Allocate memory on device for arrays
	CUDA_CHECK(hipMalloc(&d_S1, n * sizeof(int)));
	CUDA_CHECK(hipMalloc(&d_S1, n * sizeof(int)));
	CUDA_CHECK(hipMalloc(&d_S1, n * sizeof(int)));
	
	// Instanciate to zero (also optional in this case)
	CUDA_CHECK(hipMemset(d_S1, 0, n * sizeof(int)));
	CUDA_CHECK(hipMemset(d_S2, 0, n * sizeof(int)));
	CUDA_CHECK(hipMemset(d_S3, 0, n * sizeof(int)));
	
	// Copy CPU arrays to GPU (device) arrays
	CUDA_CHECK(hipMemcpy(d_S1, h_S1, n * sizeof(int), hipMemcpyHostToDevice)); // without using macro shortcut
	CUDA_CHECK(hipMemcpy(d_S1, h_S1, n * sizeof(int), H2D)); // using macro shortcut
	CUDA_CHECK(hipMemcpy(d_S1, h_S1, n * sizeof(int), H2D));
	
	// launch kernels

	int TPB = 32;
	int nblocks = ceil((float)n/TPB); 
	sum<<<nblocks, TPB>>>(d_S1, d_S2, d_S3, n);
	CUDA_CHECK_ERROR();
	
	// Retrieve result to host in S1
	CUDA_CHECK(hipMemcpy(h_S1, d_S1, n * sizeof(int), D2H));
	
	// Synchronize host and device
	CUDA_CHECK(hipDeviceSynchronize());	
	
	
	return 0;
}
