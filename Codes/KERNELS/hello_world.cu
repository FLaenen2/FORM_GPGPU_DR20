#include "hip/hip_runtime.h"
#include <stdio.h>
#include "../COMMON/commons.cuh"

__global__ void hello_world(void){   
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	printf("Hello from thread %d (global index %d) in block %d\n", threadIdx.x, i, blockIdx.x);
}

int main(int argc, char **argv){
	hello_world<<<2, 5>>>();
	SYNCGPU();
	return 0;
}