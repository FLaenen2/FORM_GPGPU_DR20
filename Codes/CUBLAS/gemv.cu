#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hipblas.h>
#include "../COMMON/common.h"

// Computes the product alpha A x + beta y = y
// A is a matrix of size M x N, x is  a vector of size N, y is a vector of size M, and alpha and beta are scalars

#define N 500
// N is number of columns
#define M 600
// M is number of rows

int main(int argc, char **argv)
{
    hipblasHandle_t handle;
    //hipblasStatus_t status;
    float *d_x, *d_A, *d_y;
    
    // Make place on CPU memory for matrix and vector. 
    float *h_x = new float[N]; 
    float *h_y = new float[M]; 
    float *h_A = new float[M*N];

    // Initialize cublas to perform operations
    //CHECK_CUBLAS(hipblasCreate(&handle));

    CHECK_CUBLAS(hipblasCreate(&handle));

    // The long way to check for error
/*	if (status != HIPBLAS_STATUS_SUCCESS)
    {
        fprintf(stderr, "!!!! CUBLAS initialization error\n");
        return EXIT_FAILURE;
    }*/

    // Create vector and matrix on host side
	for (int j = 0; j < N; j++){
	    h_x[j] = 1;
	    for (int i = 0; i < M; i++){
		h_A[j*M + i] = i;
	    }
    }

    // Allocate memory on device for the vector and the matrix
    //CHECK(hipMalloc(&d_x, N * sizeof(float)));
    CHECK(hipMalloc(&d_x, N * sizeof(float)));
    CHECK(hipMalloc(&d_y, M * sizeof(float)));
    CHECK(hipMemset(d_y, 0, M * sizeof(float)));
    CHECK(hipMalloc(&d_A, M * N * sizeof(float)));

    // Copy the host vector to device vector. Use macro for error checking.
	CHECK_CUBLAS(hipblasSetVector(N, sizeof(h_A[0]), h_x, 1, d_x, 1));
    // Copy the host matrix to device matrix. M is the leading dimension in column major format ( = number of rows).
    CHECK_CUBLAS(hipblasSetMatrix(M, N, sizeof(h_A[0]), h_A, M, d_A, M));
    
    // Note : the following is equivalent
    //CHECK_CUBLAS(hipblasSetVector(M * N, sizeof(h_A[0]), h_A, 1, d_A, 1));

    float alpha = 1;
    float beta = 0;
	
	// Perform the matrix-vector multiplication
    CHECK_CUBLAS(hipblasSgemv(handle, HIPBLAS_OP_N, M, N, &alpha, d_A, M, d_x, 1, &beta, d_y, 1));

	// Get the resulting vector
    CHECK_CUBLAS(hipblasGetVector(M, sizeof(h_y[0]), d_y, 1, h_y, 1));

    for (int i = 0; i < M; i+=max(1,N/100)){
    	printf("[%d] %g\n", i, h_y[i]);
    }

    // Release cublas handle
    CHECK_CUBLAS(hipblasDestroy(handle));

}
