/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>



/**
 * This macro checks return value of the CUDA runtime call and exits
 * the application if the call failed.
 */
#define CUDA_CHECK_RETURN(value) {											\
	hipError_t _m_cudaStat = value;										\
	if (_m_cudaStat != hipSuccess) {										\
		fprintf(stderr, "Error %s at line %d in file %s\n",					\
				hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
		exit(1);															\
	} }
#define N 100

__global__ void kernel_1() { double sum = 0.0;
for (int i = 0; i < N; i++) {
sum = sum + tan(0.1) * tan(0.1); }
}
__global__ void kernel_2() { double sum = 0.0;
for (int i = 0; i < N; i++) {
sum = sum + tan(0.1) * tan(0.1); }
}
__global__ void kernel_3() { double sum = 0.0;
for (int i = 0; i < N; i++) {
sum = sum + tan(0.1) * tan(0.1); }
}


/**
 * Host function that prepares data array and passes it to the CUDA kernel.
 */
int main(void) {

	int n_streams = 3;
	hipStream_t *streams = (hipStream_t *)malloc(n_streams * sizeof(hipStream_t));
	for (int i = 0 ; i < n_streams; i++) {
	hipStreamCreate(&streams[i]);
	}
	dim3 block(1);
	dim3 grid(1);
	for (int i = 0; i < n_streams; i++) {
		kernel_1<<<grid, block, 0, streams[i]>>>();
	//}
		//for (int i = 0; i < n_streams; i++) {
		kernel_2<<<grid, block, 0, streams[i]>>>();
		//}
		//for (int i = 0; i < n_streams; i++) {
		kernel_3<<<grid, block, 0, streams[i]>>>();
	}
printf("done\n");

	CUDA_CHECK_RETURN(hipDeviceSynchronize());	// Wait for the GPU launched work to complete
//	CUDA_CHECK_RETURN(cudaGetLastError());

	CUDA_CHECK_RETURN(hipDeviceReset());

	return 0;
}
