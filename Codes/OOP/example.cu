#include "hip/hip_runtime.h"
#include "../COMMON/commons.cuh"
#include <stdio.h>

#define N 100

class GpuClass{

	public:
		// Variables
		int arr[N];
		GpuClass(void){
			for (int i = 0; i < N; i++){
				arr[i] = i;
			}
		}
	
		// Methods
	//	__global__ void sayHello(void){ printf("hello from global function from class !\n");} // this is forbidden
		__device__  int printNumb(int i){return arr[i];}
		
};

__global__ void testClass(GpuClass obj){
	printf("Returned value : %d\n", obj.printNumb(21));
}

int main(int argc, char **argv)
{

	GpuClass myObj;
	myObj.printNumb(2); // this is forbidden if method not declared __host__
//	myObj.sayHello<<<1,1>>>();
	testClass<<<1, 1>>>(myObj);
	SYNCGPU();
	return 0;
}
