#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
#include "../COMMON/commons.cuh"
#include <stdio.h>
#include <thrust/random/normal_distribution.h>
#include <thrust/random/linear_congruential_engine.h>
#include <thrust/count.h>
#include <thrust/remove.h>
#include <thrust/sort.h>

#define N 1000000

template<typename T>
  struct square_value
  {
    __host__ __device__ T operator()(const T &x) const
    {
      return x * x;
    }
  };
  
template<typename T>
struct is_gt
  {
  	float th;
  	is_gt(float _th) : th(_th) {};
    __host__ __device__
    bool operator()(const T x)
    {
      return abs(x) > th;
    }
  };
  
int main(int argc, char **argv){

    // Create a device_vector with a given number of elements and value
    thrust::host_vector<float> h1(N, 0.);
    thrust::host_vector<float> h2(N, 0.);
    
    // Create a generator
    thrust::minstd_rand rng(clock());
	thrust::random::normal_distribution<float> dist1(0, 1);
	thrust::random::normal_distribution<float> dist2(0, 2);
	
	// Fill host vector
	for (int i = 0; i < N; i++){
		h1[i] = dist1(rng);
		h2[i] = dist2(rng);
	}
	
	// Copy to device
	thrust::device_vector<float> v1(h1);
	thrust::device_vector<float> v2(h2);
	//print_dev<<<1,1>>>((v1.data()).get(), 100);
	
	float mean1 = thrust::reduce(v1.begin(), v1.end()) / N;
	float var1 = thrust::transform_reduce(v1.begin(), v1.end(), square_value<float>(), 0.f, thrust::plus<float>()) / N;
	float mean2 = thrust::reduce(v2.begin(), v2.end()) / N;
	float var2 = thrust::transform_reduce(v2.begin(), v2.end(), square_value<float>(), 0.f, thrust::plus<float>()) / N;

	printf("Mean1 : %g\nVariance1 : %g\n", mean1, var1);
	printf("Mean2 : %g\nVariance2 : %g\n", mean2, var2);
	
	float threshold = 1.;
	int result = thrust::count_if(v1.begin(), v1.end(), is_gt<float>(threshold));
	v1.erase(thrust::remove_if(v1.begin(), v1.end(), is_gt<float>(threshold)), v1.end());
	printf("Fraction of elements whose absolute value is greater than %g : %g\nFraction of original size after erase : %g\n", threshold , (float)result / N, (float)v1.size() / N);

	// SORTING
	bool is_sorted = thrust::is_sorted(v1.begin(), v1.end());
	printf("Is sorted ? %d\n", is_sorted);	
	thrust::sort(v1.begin(), v1.end());
	is_sorted = thrust::is_sorted(v1.begin(), v1.end());
	printf("Is sorted ? %d\n", is_sorted);
    
    SYNCGPU();
    return 0;
    
}