#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
//#include <thrust/sort.h>
//#include <thrust/copy.h>
//#include <thrust/sequence.h>
//#include <thrust/random.h>
//#include <thrust/random/normal_distribution.h>

#include <algorithm>
#include <time.h>
//#include <limits.h>
#include "../COMMON/commons.cuh"
#include <stdio.h>

#define N 1000
using namespace thrust;

/*template<typename T>
struct Rands{
	
	Rands<T>(void){
		rng = default_random_engine(clock()); 
		dist = normal_distribution<float>(0, 1);
	};
	default_random_engine rng;
	normal_distribution<float> dist;
	__host__ __device__ T operator()(T &a){
		return dist(rng);
	};

};*/


int main(int argc, char **argv){
	
//	thrust::default_random_engine rng(clock());
	//thrust::normal_distribution<float> dist1(0, 1);
	//thrust::normal_distribution<float> dist2(0, 2);
	// Declare device vectors and initialize elements with 0
	//host_vector<float> h_v1(N, 0);
//	host_vector<float> h_v2(N);
//	for (int i = 0; i < h_v1.size(); i++){
	//	h_v1[i] = dist1(rng);
//		h_v2[i] = dist2(rng);
//	}

	device_vector<float> d_v1(10); 
	//device_vector<float> d_v2(N); 
	//thrust::copy(h_v1.begin(), h_v2.end(), d_v1.begin());
//	d_v1 = h_v1;
//	d_v2 = h_v2;
//	float res = thrust::reduce(d_v1.begin(), d_v1.end(), 0, thrust::plus<float>());
//	printf("Mean 1 : %g\n", res);
//	thrust::sort(d_v1.begin(), d_v1.end());
//	bool is_sorted = thrust::is_sorted(d_v1.begin(), d_v1.end()); 
//	printf("is d_v1 sorted : %d", is_sorted);


//	Rands<float> op(); 
//	transform(d_v1.begin(), d_v1.end(), d_v1.begin(), op);
//	print_dev<<<1, 1>>>(thrust::raw_pointer_cast(d_v1.data()), 100);
	CUDA_CHECK(hipDeviceSynchronize());
	//CUDA_CHECK(hipDeviceReset());
	return 0;

}
