#include "hip/hip_runtime.h"
#include "../common/common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * This example demonstrates the impact of misaligned reads on performance by
 * forcing misaligned reads to occur on a PREC*.
 */
#define PREC float
void checkResult(PREC *hostRef, PREC *gpuRef, const int N)
{
    double epsilon = 1.0E-8;
    bool match = 1;

    for (int i = 0; i < N; i++)
    {
        if (abs(hostRef[i] - gpuRef[i]) > epsilon)
        {
            match = 0;
            printf("different on %dth element: host %f gpu %f\n", i, hostRef[i],
                    gpuRef[i]);
            break;
        }
    }

    if (!match)  printf("Arrays do not match.\n\n");
}

void initialData(PREC *ip,  int size)
{
    for (int i = 0; i < size; i++)
    {
        ip[i] = (PREC)( rand() & 0xFF ) / 100.0f;
    }

    return;
}


void sumArraysOnHost(PREC *A, PREC *B, PREC *C, const int n, int offset)
{
    for (int idx = offset, k = 0; idx < n; idx++, k++)
    {
        C[k] = A[idx] + B[idx];
    }
}

__global__ void warmup(PREC *A, PREC *B, PREC *C, const int n, int offset)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int k = i + offset;

    if (k < n) C[i] = A[k] + B[k];
}

__global__ void readOffset(PREC *A, PREC *B, PREC *C, const int n,
                           int offset)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int k = i + offset;

    if (k < n) C[i] = A[k] + B[k];
}

int main(int argc, char **argv)
{
    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("%s starting reduction at ", argv[0]);
    printf("device %d: %s ", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    // set up array size
    int nElem = 1 << 24; // total number of elements to reduce
    printf(" with array size %d\n", nElem);
    size_t nBytes = nElem * sizeof(PREC);

    // set up offset for summary
    int blocksize = 512;
    int offset = 0;

    if (argc > 1) offset    = atoi(argv[1]);

    if (argc > 2) blocksize = atoi(argv[2]);

    // execution configuration
    dim3 block (blocksize, 1);
    dim3 grid  ((nElem + block.x - 1) / block.x, 1);

    // allocate host memory
    PREC *h_A = (PREC *)malloc(nBytes);
    PREC *h_B = (PREC *)malloc(nBytes);
    PREC *hostRef = (PREC *)malloc(nBytes);
    PREC *gpuRef  = (PREC *)malloc(nBytes);

    //  initialize host array
    initialData(h_A, nElem);
    memcpy(h_B, h_A, nBytes);

    //  summary at host side
    sumArraysOnHost(h_A, h_B, hostRef, nElem, offset);

    // allocate device memory
    PREC *d_A, *d_B, *d_C;
    CHECK(hipMalloc((PREC**)&d_A, nBytes));
    CHECK(hipMalloc((PREC**)&d_B, nBytes));
    CHECK(hipMalloc((PREC**)&d_C, nBytes));

    // copy data from host to device
    CHECK(hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_B, h_A, nBytes, hipMemcpyHostToDevice));

    //  kernel 1:
    double iStart = seconds();
    warmup<<<grid, block>>>(d_A, d_B, d_C, nElem, offset);
    CHECK(hipDeviceSynchronize());
    double iElaps = seconds() - iStart;
    printf("warmup     <<< %4d, %4d >>> offset %4d elapsed %f sec\n", grid.x,
           block.x, offset, iElaps);
    CHECK(hipGetLastError());

    int nreps = 1;
    float total = 0.;
    for (int i = 0; i < nreps; i++){
	CHECK(hipMemset(d_C, 0, nElem * sizeof(PREC)));
	iStart = seconds();
	readOffset<<<grid, block>>>(d_A, d_B, d_C, nElem, offset);
	CHECK(hipDeviceSynchronize());
	iElaps = seconds() - iStart;
	total += iElaps; 
	
    }
    printf("readOffset <<< %4d, %4d >>> offset %4d elapsed %f sec\n", grid.x,
           block.x, offset, total/nreps);
    CHECK(hipGetLastError());

    // copy kernel result back to host side and check device results
    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));
    checkResult(hostRef, gpuRef, nElem - offset);

    // free host and device memory
    CHECK(hipFree(d_A));
    CHECK(hipFree(d_B));
    CHECK(hipFree(d_C));
    free(h_A);
    free(h_B);

    // reset device
    CHECK(hipDeviceReset());
    return EXIT_SUCCESS;
}
