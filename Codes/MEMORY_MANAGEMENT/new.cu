#define N 100000
#include "../COMMON/commons.cuh"
int main(int argc, char **argv){

    float *d_arr;
    // Memory allocation
    CUDA_CHECK(hipMalloc(&d_arr, N * sizeof(float)));
    // Memory set
    CUDA_CHECK(hipMemset(d_arr, 0, N * sizeof(float)));
    
    float *h_arr = new float[N]();
    for (int i = 0; i < N; i++){
	h_arr[i] = (float) i;
    }

    // Memory Copy to GPU (destination, source, size, direction)
    CUDA_CHECK(hipMemcpy(d_arr, h_arr, N * sizeof(float), hipMemcpyHostToDevice));
    // Works also thanks to UVA
    CUDA_CHECK(hipMemcpy(d_arr, h_arr, N * sizeof(float), hipMemcpyDefault));

    //test<<<1,1>>>(h_arr);
    
    CUDA_CHECK(hipMemcpy(h_arr, d_arr, N * sizeof(float), D2H));

    CUDA_CHECK(hipFree(d_arr));

    CUDA_CHECK(hipDeviceSynchronize());

    

return 0;
}
