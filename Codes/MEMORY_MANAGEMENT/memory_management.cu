#include "hip/hip_runtime.h"

// Illustrates the main functions used in CUDA to allocate and free memory, and copy it between host and device

#include "../COMMON/commons.cuh"
#define N 10000
#define H2D hipMemcpyHostToDevice

int main(int argc, char **argv){
 
    // Create a host vector in host / cpu memory
    float *hv = new float[N];
    
    // Initialize it 
    for (int i = 0; i < N; i++){
	hv[i] = i * sqrt(2);
    }
    
    // Declare a pointer that will reside on the device
    float *dv;    
    
    // Allocate memory using this last pointer. Note that the function requires the adress of the pointer
    CUDA_CHECK(hipMalloc(&dv, N * sizeof(float)));

    // Initialize it to 0 (not necessary if will be copied over but good practice)
    CUDA_CHECK(hipMemset(dv, 0, N * sizeof(float)));
    
    // Copy the array from CPU to GPU (host to device)
    CUDA_CHECK(hipMemcpy(dv, hv, N * sizeof(float), hipMemcpyHostToDevice)); // hipMemcpyDefault works also
    // Illustration 
    CUDA_CHECK(hipMemcpy(dv, hv, N * sizeof(float), hipMemcpyDefault)); // works also
    CUDA_CHECK(hipMemcpy(dv, hv, N * sizeof(float), H2D)); // using a macro fro shortcut

    // Perform some operations on it
    // ...
    // Get the data back to the host
    CUDA_CHECK(hipMemcpy(hv, dv, N * sizeof(float), hipMemcpyDeviceToHost));
    
    // Free useless device memory
    CUDA_CHECK(hipFree(dv));

    // Synchronize 
    CUDA_CHECK(hipDeviceSynchronize());

    printf("Success\n");
    return 0;
}


