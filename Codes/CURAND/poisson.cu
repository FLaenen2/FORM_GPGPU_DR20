#include <stdio.h>
#include <hiprand.h>
#include "../COMMON/commons.cuh"
#include <thrust/device_ptr.h>
#include <thrust/reduce.h>

// Generates a random vector from a Poisson distribution from host side with parameter lamba. 
// Then computes the mean (should be equal to lambda).

#define N 100000

int main(int argc, char **argv)
{
	unsigned int lambda = 3;
	hiprandGenerator_t gen;
	unsigned int *h_v = new unsigned int[N];
	unsigned int *d_v;

	// Allocate device memory
	CUDA_CHECK(hipMalloc(&d_v, N * sizeof(unsigned int)));

	// Create a hiprand generator (several generators exist, let's just take the default)
	CURAND_CHECK(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));

	// Set a seed
	CURAND_CHECK(hiprandSetPseudoRandomGeneratorSeed(gen, 1));
		
	// Generate the random numbers on the device vector
	printf("Generating poisson distribution with parameter : %d\n", lambda);
	CURAND_CHECK(hiprandGeneratePoisson(gen, d_v, N, lambda));
	
	// Compute and print the mean
	thrust::device_ptr<unsigned int> t_v = thrust::device_pointer_cast(d_v);
	float mean = (float) thrust::reduce(t_v, t_v + N);
	mean /= N;
	printf("Mean : %g\n", mean);

	// Optionally copy the vector back to the CPU
	CUDA_CHECK(hipMemcpy(h_v, d_v, N * sizeof(unsigned int), hipMemcpyDeviceToHost));
	
	// Release the hiprand resource
	CURAND_CHECK(hiprandDestroyGenerator(gen));


	
   return 0;
}
