
#include <hip/hip_runtime.h>
#include <stdio.h>
__device__ float giveFloat(void);
__device__ int dev(){ return 2;}
__global__ void foo(void){
   int a =  dev(); // calling the device function
    printf("a = %d\n", a);
    float b = giveFloat();
}

void wrapper(void){
    foo<<<1,1>>>(); // calling the global function
    hipDeviceSynchronize();
}
