
#include <hip/hip_runtime.h>
__global__ void add(double *a, double *b, float c, int size){

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < size){
        a[i] += b[i] * c;
    }
    
}