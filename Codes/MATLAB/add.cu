
#include <hip/hip_runtime.h>
__global__ void add(double *a, double *b, const float c, const int size){

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < size){
        a[i] += b[i] * c;
    }
    
}